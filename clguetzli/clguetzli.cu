
#include <hip/hip_runtime.h>
#ifdef __HIPCC__
//#ifdef __OPENCL_VERSION__
__device__ int get_global_id(int dim)
{
    switch (dim)
    {
    case 0:
        return threadIdx.x;
    case 1:
        return threadIdx.y;
    case 2:
        return threadIdx.z;
    default:
        return threadIdx.x;
    }
}
#endif


__global__ void clScaleImageEx(float *img, double scale)
{
    const int i = get_global_id(0);
    img[i] *= scale;
}